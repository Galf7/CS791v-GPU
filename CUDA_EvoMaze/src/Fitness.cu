#include "hip/hip_runtime.h"

#include "Fitness.h"
#include <unistd.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ bool Debug = true;

struct Coord {
public:
	int x;
	int y;
};


__device__ Coord* GetRegionTiles(int startX,int startY,int mapSize,int **map){
	if(Debug && threadIdx.x == 0){
		printf("Getting Region Tiles\n");
	}
	Coord *tiles;
	tiles = new Coord[mapSize*mapSize];
	int** mapFlags;
	int tileType = map[startX][startY];
	//init mapFlags
	mapFlags = new int*[mapSize];
	for(int flags = 0; flags < mapSize; flags++){
		mapFlags[flags] = new int[mapSize];
		for(int init = 0; init < mapSize; init++){
			mapFlags[flags][init] = 0;
		}
	}

	tiles[0].x = startX;
	tiles[0].y = startY;
	mapFlags [startX][startY] = 1;

	int iter = 0;
	int cursor = 0;
	if(Debug && threadIdx.x == 0){
		printf("%d, %d; ",tiles[iter].x,tiles[iter].y);
	}
	while (tiles[cursor].x != -1 && cursor < mapSize*mapSize) {

		for (int x = tiles[cursor].x - 1; x <= tiles[cursor].x + 1; x++) {
			for (int y = tiles[cursor].y - 1; y <= tiles[cursor].y + 1; y++) {
				if ((x >= 0  && x < mapSize && y >= 0  && y < mapSize) && (y == tiles[cursor].y || x == tiles[cursor].x)) {
					if (mapFlags[x][y] == 0 && map[x][y] == tileType) {
						mapFlags[x][y] = 1;
						iter++;
						tiles[iter].x = x;
						tiles[iter].y = y;
						if(Debug && threadIdx.x == 0){
							printf("%d, %d; ",tiles[iter].x,tiles[iter].y);
						}
					}
				}
			}
		}
		if(iter == cursor && iter < mapSize*mapSize){
			iter++;
			tiles[iter].x = -1;
			tiles[iter].y = -1;
			cursor++;
		}
		else{
			cursor++;
		}
	}

	for(int i = 0; i < mapSize; i++){
		delete(mapFlags[i]);
	}
	delete(mapFlags);

	return tiles;
}

__device__ Coord** GetRegions(int **map, int mapSize, int startX, int startY, int tileType){
	if(Debug && threadIdx.x == 0){
		printf("Getting Regions\n");
	}
	Coord** regions;
	int** mapFlags;

	//init regions
	regions = new Coord*[mapSize*mapSize];
	//init mapFlags
	mapFlags = new int*[mapSize];
	for(int flags = 0; flags < mapSize; flags++){
		mapFlags[flags] = new int[mapSize];
		for(int init = 0; init < mapSize; init++){
			mapFlags[flags][init] = 0;
		}
	}

	int cursor = 0;
	int iter = 0;
	int roomNum = 0;
	for (int row = 0; row < mapSize; row ++) {
		for (int col = 0; col < mapSize; col ++) {
			if (mapFlags[row][col] == 0 && map[row][col] == tileType) {
				Coord *newRegion = GetRegionTiles(row,col,mapSize,map);
				roomNum = 0;
				while(newRegion[roomNum].x != -1){
					roomNum++;
				}
				regions[cursor] = new Coord[roomNum];
				for(int i = 0; i < roomNum; i++){
					regions[cursor][i].x = newRegion[i].x;
					regions[cursor][i].y = newRegion[i].y;
				}
				cursor++;
				iter = 0;
				while(newRegion[iter].x != -1 && iter < mapSize*mapSize){
					mapFlags[newRegion[iter].x][newRegion[iter].y] = 1;
					if(Debug && threadIdx.x == 0){
						printf("%d, %d; ",newRegion[iter].x,newRegion[iter].y);
					}
					iter++;
				}
				if(Debug && threadIdx.x == 0){
					printf("\n");
				}
				delete(newRegion);
			}
		}
	}
	if(cursor < mapSize*mapSize){
		Coord* newRegion = new Coord[1];
		regions[cursor] = newRegion;
		regions[cursor][0].x = -1;
		regions[cursor][0].y = -1;
		delete(newRegion);
	}
	for(int i = 0; i < mapSize; i++){
		delete(mapFlags[i]);
	}
	delete(mapFlags);
	return regions;
}

__device__ void MakePassage(int **map, Coord tileA, Coord tileB){
	if(Debug && threadIdx.x == 0){
		printf("Making Passage\n");
	}
	int cursor = 0;
	int target = 0;
	int prevX = 0;
	cursor = tileA.x;
	target = tileB.x;

	if ((tileA.x - tileB.x) * (tileA.x - tileB.x) > 0) {
		if (tileA.x > tileB.x) {
			while (cursor > target) {
				cursor--;
				map [cursor] [tileA.y] = 0;
			}
		} else {
			while (cursor < target) {
				cursor++;
				map [cursor] [tileA.y] = 0;
			}
		}
	}
	prevX = cursor;
	if ((tileA.y - tileB.y) * (tileA.y - tileB.y) > 0) {
		if (tileA.y > tileB.y) {
			cursor = tileA.y;
			target = tileB.y;
			while (cursor > target) {
				cursor--;
				map [prevX] [cursor] = 0;
			}
		} else {
			cursor = tileA.y;
			target = tileB.y;
			while (cursor < target) {
				cursor++;
				map [prevX] [cursor] = 0;
			}
		}
	}
}

__device__ void ConnectClosestRooms(Coord **rooms, int **map, int mapSize){
	if(Debug && threadIdx.x == 0){
		printf("Connect Closest Rooms\n");
	}
	int roomNum = 0;
	bool **connected;
	bool *accessibleToStart;

	if(Debug && threadIdx.x == 0){
		printf("Finding Room total\n");
	}
	while(rooms[roomNum][0].x != -1 && roomNum < mapSize*mapSize){
		roomNum++;
	}

	if(Debug && threadIdx.x == 0){
		printf("setting connected tables\n");
	}
	connected = new bool*[roomNum];
	accessibleToStart = new bool[roomNum];
	for(int row = 0; row < roomNum; row++){
		connected[row] = new bool[roomNum];
		accessibleToStart[row] = false;
		for(int col = 0; col < roomNum; col++){
			connected[row][col] = false;
			if(row == col){
				connected[row][col] = true;
			}
		}
	}
	accessibleToStart[0] = true;
	//while there are disconnected rooms
	int access = 1;
	Coord bestTileA;
	Coord bestTileB;
	int bestRoom1;
	int bestRoom2;
	bool possibleConnection = false;
	int bestDistance = 0;

	if(Debug && threadIdx.x == 0){
		printf("Connecting Rooms\n");
	}
	while(access < roomNum){
		possibleConnection = false;
		bestDistance = 0;
		bestRoom1 = 0;
		bestRoom2 = 0;
		//find two nearest rooms
		int tileA = 0;
		int tileB = 0;
		for(int room = 0; room < roomNum; room++){
			for(int room2 = 0; room2 < roomNum; room2++){
				if(!connected[room][room2] && !connected[room2][room]){
					tileA = 0;
					while(rooms[room][tileA].x != -1 && tileA < mapSize*mapSize){
						tileB = 0;
						while(rooms[room2][tileB].x != -1 && tileB < mapSize*mapSize){
							int distance = (rooms[room][tileA].x - rooms[room2][tileB].x) * (rooms[room][tileA].x - rooms[room2][tileB].x);
							distance = distance + (rooms[room][tileA].y - rooms[room2][tileB].y) * (rooms[room][tileA].y - rooms[room2][tileB].y);

							if(Debug && threadIdx.x == 0){
								printf("Rooms: %d, %d;\n",room,room2);
								printf("Tile: %d, %d;\n",rooms[room][tileA].x,rooms[room][tileA].y);
								printf("Tile: %d, %d;\n",rooms[room2][tileB].x,rooms[room2][tileB].y);
							}
							if(distance < bestDistance || !possibleConnection){
								bestDistance = distance;
								possibleConnection = true;
								bestTileA.x = rooms[room][tileA].x;
								bestTileA.y = rooms[room][tileA].y;
								bestTileB.x = rooms[room2][tileB].x;
								bestTileB.y = rooms[room2][tileB].y;
								bestRoom1 = room;
								bestRoom2 = room2;
							}
							tileB++;
						}
						tileA++;
					}
				}
			}
		}
		//connect those rooms
		if(Debug && threadIdx.x == 0){
			//printf("%d, %d;\n",bestTileA.x,bestTileA.y);
			//printf("%d, %d;\n",bestTileB.x,bestTileB.y);
		}
		MakePassage(map,bestTileA,bestTileB);
		connected[bestRoom1][bestRoom2] = true;
		connected[bestRoom2][bestRoom1] = true;
		//check if accessible to start
		if((accessibleToStart[bestRoom1] && !accessibleToStart[bestRoom2])){
			accessibleToStart[bestRoom2] = true;
			access++;
			for(int iter = 0; iter < roomNum; iter++){
				if(connected[bestRoom2][iter] && !accessibleToStart[iter]){
					accessibleToStart[iter] = true;
					access++;
				}
			}
		}
		else if((!accessibleToStart[bestRoom1] && accessibleToStart[bestRoom2])){
			accessibleToStart[bestRoom1] = true;
			access++;
			for(int iter = 0; iter < roomNum; iter++){
				if(connected[bestRoom1][iter] && !accessibleToStart[iter]){
					accessibleToStart[iter] = true;
					access++;
				}
			}
		}
	}
	delete(accessibleToStart);
	for(int i = 0; i < roomNum; i++){
		delete(connected[i]);
		delete(rooms[i]);
	}
	delete(rooms);
	delete(connected);
}

__device__ int GetNeighbors(int **map, int x, int y, int mapSize){
	int neighbors = 0;
	int row = x-1;
	int col = y-1;
	int rowMax = x+1;
	int colMax = y+1;

	//check for out of bounds
	if(row < 0){
		row = 0;
	}
	if (rowMax >= mapSize){
		rowMax = mapSize-1;
	}
	if (colMax >= mapSize){
		colMax = mapSize-1;
	}
	//get neighbors
	for(; row < rowMax; row++){
		for(col = y-1; col < colMax; col++){
			if(col < 0){
				col = 0;
			}
			if(!(row == x && col == y) && map[row][col] == 1){
				neighbors++;
			}
		}
	}

	return neighbors;
}

__device__ void RunCA(int **map, int mapSize, float* rules, unsigned int seed){
	int **CAmap;
	int maxCA = 50;

	CAmap = new int*[mapSize];
	for(int iter = 0; iter < mapSize; iter++){
		CAmap[iter] = new int[mapSize];
	}
	hiprandState_t state;

	  /* we have to initialize the state */
	  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
	              0, /* the sequence number is only important with multiple cores */
	              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	              &state);

	for(int caIter = 0; caIter < maxCA; caIter++){
		for(int x = 0; x < mapSize; x++){
			for(int y = 0; y < mapSize; y++){
				//rules stuff here
				float rand = hiprand(&state)%127;
				if(map[x][y] < 1 && rules[GetNeighbors(map,x,y,mapSize)] > rand){
					CAmap[x][y] = 1;
				}
				else if(map[x][y] > 0 && rules[GetNeighbors(map,x,y,mapSize)+9] > rand){
					CAmap[x][y] = 0;
				}
			}
		}
		//set map equal to CAmap
		for(int x = 0; x < mapSize; x++){
			for(int y = 0; y < mapSize; y++){
				map[x][y] = CAmap[x][y];
				}
		}
	}
	for(int i = 0; i < mapSize; i++){
		delete(CAmap[i]);
	}
	delete(CAmap);
}

/*
  This is the function that each thread will execute on the GPU. The
  fact that it executes on the device is indicated by the __global__
  modifier in front of the return type of the function. After that,
  the signature of the function isn't special - in particular, the
  pointers we pass in should point to memory on the device, but this
  is not indicated by the function's signature.
 */
__global__ void GetFitnesses(float **population, float *fitnesses, int popSize,int chromSize,int mapSize, unsigned int seed) {
	int chrom = (threadIdx.x)%popSize;
	int** map;
	//printf("%d, ",chrom);
	//init map
	map = new int*[mapSize];
	for(int iter = 0; iter < mapSize; iter++){
		map[iter] = new int[mapSize];
		for(int curs = 0; curs < mapSize; curs++){
			map[iter][curs] = 0;
		}
	}

	//build maze pattern
	RunCA(map,mapSize,population[chrom],seed);
	//clear start and end tiles
	map[0][0] = 0;
	map[mapSize-1][mapSize-1] = 0;
	if(threadIdx.x == 0 && Debug){
		for(int x = 0; x < mapSize; x++){
			for(int y = 0; y < mapSize; y++){
				printf("%d",map[x][y]);
			}
			printf("\n");
		}
	}
	//find disconnected rooms
	Coord **rooms = GetRegions(map,mapSize,0,0,0);

	//connect disconnected rooms
	if(threadIdx.x == 0){
	ConnectClosestRooms(rooms,map,mapSize);
	}
	/*int roomNum = 0;
	while(rooms[roomNum][0].x != -1){
		roomNum++;
	}
	for(int temp = 0; temp < roomNum; temp++){
		delete(rooms[temp]);
	}
	delete(rooms);*/

	//Check fitness of maze

	for(int i = 0; i < mapSize; i++){
		delete (map[i]);
	}
	delete(map);
}


__global__ void mult(int size,int** a, int** b, int** c, int** d, int** e) {
	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;
	int x, y;
	/*for(int id_x = blockIdx.x * blockDim.x + threadIdx.x; id_x < size; id_x += stride_x){
		for(int id_y = blockIdx.y * blockDim.y + threadIdx.y; id_y < size;	id_y += stride_y){
			c[id_x][id_y] = a[id_x][id_y] * b[id_y][id_x];
		}
	}*/
	for(int j = (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x
			+ (blockIdx.x * blockDim.x + threadIdx.x);
			j < size*size; j += stride_x * stride_y){
		x = j/size;
		y = j%size;
		c[x][y] = (a[x][y] * b[y][x]) + (d[x][y] * e[y][x]);
	}
}
